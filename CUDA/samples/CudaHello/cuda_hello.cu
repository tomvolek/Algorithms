
// System includes
#include <stdio.h>
#include <assert.h>
#include <unistd.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


// kernel definition 

__global__ void veectorAdd(int * a, int *b, int *c) { 

    int i = threadIdx.x;
    c[i] = a[i] + b[i];
    printf("inside kernel code\n") ; 
    return;

}

int main() { 
    int a[]= { 1,2,3};
    int b[] = {4,5,6};
    int c[sizeof(a) /sizeof(int)] = {0};

int *cudaA = 0; 
int *cudaB = 0;
int *cudaC = 0; 

hipMalloc(&cudaA, sizeof(a));
hipMalloc(&cudaB, sizeof(a));
hipMalloc(&cudaC, sizeof(a));

//allcoate memory
hipMemcpy(cudaA,a,sizeof(a), hipMemcpyHostToDevice);
hipMemcpy(cudaB,b,sizeof(b), hipMemcpyHostToDevice);
hipMemcpy(cudaC,c,sizeof(c), hipMemcpyHostToDevice);

printf("Inside main\n");

    veectorAdd<<<1,sizeof(a) /sizeof(int)>>>(cudaA,cudaB, cudaC);
    hipMemcpy (c,cudaC,sizeof(c),hipMemcpyDeviceToHost);
    return 0 ;
}





